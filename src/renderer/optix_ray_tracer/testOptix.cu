#include "hip/hip_runtime.h"

#include "device_math_vec.hpp"
#include "optix_parameters.hpp"
#include <optix_device.h>

namespace VTX
{
	namespace Renderer
	{
		extern "C" __constant__ Optix::LaunchParameters params;

		static __forceinline__ __device__ void setPayload( const float3 & p )
		{
			optixSetPayload_0( float_as_int( p.x ) );
			optixSetPayload_1( float_as_int( p.y ) );
			optixSetPayload_2( float_as_int( p.z ) );
		}

		static __forceinline__ __device__ float3 getPayload()
		{
			return make_float3( int_as_float( optixGetPayload_0() ),
								int_as_float( optixGetPayload_1() ),
								int_as_float( optixGetPayload_2() ) );
		}

		__forceinline__ __device__ uchar4 make_color( const float3 & c )
		{
			return make_uchar4( static_cast<uint8_t>( clamp( c.x, 0.f, 1.f ) * 255.f ),
								static_cast<uint8_t>( clamp( c.y, 0.f, 1.f ) * 255.f ),
								static_cast<uint8_t>( clamp( c.z, 0.f, 1.f ) * 255.f ),
								255u );
		}

		extern "C" __global__ void __closesthit__sphere()
		{
			Optix::HitGroupData * data = reinterpret_cast<Optix::HitGroupData *>( optixGetSbtDataPointer() );

			const float3 normal = make_float3( int_as_float( optixGetAttribute_0() ),
													  int_as_float( optixGetAttribute_1() ),
													  int_as_float( optixGetAttribute_2() ) );
			const int	 id		= optixGetPrimitiveIndex();
			const float3 & color	= params._colors[data->_spheres[ id ]._colorId];
			const float3 &rayDir = optixGetWorldRayDirection();
			const float	 radiance = fabsf( dot( rayDir, normal ) );


			setPayload( color * radiance);
		}

		extern "C" __global__ void __closesthit__cylinder()
		{
			Optix::HitGroupData * data = reinterpret_cast<Optix::HitGroupData *>( optixGetSbtDataPointer() );

			const float3   normal = make_float3( int_as_float( optixGetAttribute_0() ),
												 int_as_float( optixGetAttribute_1() ),
												 int_as_float( optixGetAttribute_2() ) );
			const int	   id	  = optixGetPrimitiveIndex();
			const float3 & color	= params._colors[data->_cylinders[ id ]._colorId];
			const float3 & rayDir	= optixGetWorldRayDirection();
			const float	   radiance = fabsf( dot( rayDir, normal ) );

			setPayload( color * radiance );
		}

		//extern "C" __global__ void __anyhit__() {}

		extern "C" __global__ void __miss__()
		{
			Optix::MissData * data	  = reinterpret_cast<Optix::MissData *>( optixGetSbtDataPointer() );
			float3			  payload = getPayload();
			setPayload( data->_colorBackground );
		}

		static __forceinline__ __device__ void trace( const OptixTraversableHandle & th,
													  const float3					 &rayOrigin,
													  const float3					 &rayDirection,
													  const float					 tMin,
													  const float					 tMax,
													  float3 *						 perRayData )
		{
			uint32_t p0, p1, p2;
			p0 = float_as_int( perRayData->x );
			p1 = float_as_int( perRayData->y );
			p2 = float_as_int( perRayData->z );

			optixTrace( th, // GAS
						rayOrigin,
						rayDirection,
						tMin,
						tMax,
						0.f, // ray time
						OptixVisibilityMask( 1 ),
						OPTIX_RAY_FLAG_NONE,
						0, // SBT offset
						0, // SBT stride
						0, // miss SBT index
						p0,
						p1,
						p2 );

			perRayData->x = int_as_float( p0 );
			perRayData->y = int_as_float( p1 );
			perRayData->z = int_as_float( p2 );
		}

		extern "C" __global__ void __raygen__()
		{
			const uint3 &id	= optixGetLaunchIndex();
			const uint3 &dim = optixGetLaunchDimensions();

			const Optix::RayGeneratorData * data
				= reinterpret_cast<Optix::RayGeneratorData *>( optixGetSbtDataPointer() );

			// camera data
			const float3 & origin = data->_camera._position;
			const float3 & front  = data->_camera._front;
			const float3 & du	  = data->_camera._du;
			const float3 & dv	  = data->_camera._dv;

			const float2 d = make_float2( float( id.x ) / float( dim.x ), float( id.y ) / float( dim.y ) ) * 2.f - 1.f;

			const float3 rayDir = normalize( du * d.x + dv * d.y + front );

			const uint32_t frameBufferId = id.x + id.y * dim.x;

			float3 normal;

			trace( params._traversable, // GAS
				   origin,
				   rayDir,
				   1e-3f, // tMin
				   1e32f, // tMax
				   &normal );

			params._frame._pixels[ frameBufferId ] = make_color( normal );
		}

		extern "C" __global__ void __intersection__sphere()
		{
			Optix::HitGroupData * data = reinterpret_cast<Optix::HitGroupData *>( optixGetSbtDataPointer() );

			// primitive data
			const int	   id	  = optixGetPrimitiveIndex();
			
			Optix::Intersection hit;
			if (data->_spheres[id].intersect(
				optixGetObjectRayOrigin(), optixGetObjectRayDirection(), optixGetRayTmin(), optixGetRayTmax(), hit))
			{
				unsigned int p0 = float_as_int( hit._normal.x );
				unsigned int p1 = float_as_int( hit._normal.y );
				unsigned int p2 = float_as_int( hit._normal.z );

				optixReportIntersection( hit._t, 0, p0, p1, p2 );
			}
		}

		extern "C" __global__ void __intersection__cylinder()
		{
			Optix::HitGroupData * data = reinterpret_cast<Optix::HitGroupData *>( optixGetSbtDataPointer() );

			// primitive data
			const int id = optixGetPrimitiveIndex();

			Optix::Intersection hit;
			if ( data->_cylinders[ id ].intersect( optixGetObjectRayOrigin(),
												 optixGetObjectRayDirection(),
												 optixGetRayTmin(),
												 optixGetRayTmax(),
												 hit ) )
			{
				unsigned int p0 = float_as_int( hit._normal.x );
				unsigned int p1 = float_as_int( hit._normal.y );
				unsigned int p2 = float_as_int( hit._normal.z );

				optixReportIntersection( hit._t, 0, p0, p1, p2 );
			}
		}
	} // namespace Renderer
} // namespace VTX
