#include "hip/hip_runtime.h"

#include "device_math_vec.hpp"
#include "optix_parameters.hpp"
#include <optix_device.h>

namespace VTX
{
	namespace Renderer
	{
		extern "C" __constant__ Optix::LaunchParameters params;

		static __forceinline__ __device__ void setPayload( const float3 & p )
		{
			optixSetPayload_0( float_as_int( p.x ) );
			optixSetPayload_1( float_as_int( p.y ) );
			optixSetPayload_2( float_as_int( p.z ) );
		}

		static __forceinline__ __device__ float3 getPayload()
		{
			return make_float3( int_as_float( optixGetPayload_0() ),
								int_as_float( optixGetPayload_1() ),
								int_as_float( optixGetPayload_2() ) );
		}

		__forceinline__ __device__ uchar4 make_color( const float3 & c )
		{
			return make_uchar4( static_cast<uint8_t>( clamp( c.x, 0.f, 1.f ) * 255.f ),
								static_cast<uint8_t>( clamp( c.y, 0.f, 1.f ) * 255.f ),
								static_cast<uint8_t>( clamp( c.z, 0.f, 1.f ) * 255.f ),
								255u );
		}

		extern "C" __global__ void __closesthit__()
		{
			Optix::HitGroupData * data = reinterpret_cast<Optix::HitGroupData *>( optixGetSbtDataPointer() );

			const float3 normal = make_float3( int_as_float( optixGetAttribute_0() ),
													  int_as_float( optixGetAttribute_1() ),
													  int_as_float( optixGetAttribute_2() ) );
			const int	 id		= optixGetPrimitiveIndex();
			const float3 color	= data->_colors[ id ];
			const float3 rayDir = optixGetWorldRayDirection();
			const float	 radiance = 0.2f + 0.8f * fabsf( dot( rayDir, normal ) );


			setPayload( color * radiance );
			//setPayload( normalize( optixTransformNormalFromObjectToWorldSpace( normal ) ) * 0.5f + 0.5f );
		}

		//extern "C" __global__ void __anyhit__() {}

		extern "C" __global__ void __miss__()
		{
			Optix::MissData * data	  = reinterpret_cast<Optix::MissData *>( optixGetSbtDataPointer() );
			float3			  payload = getPayload();
			setPayload( data->_colorBackground );
		}

		static __forceinline__ __device__ void trace( const OptixTraversableHandle & th,
													  const float3					 rayOrigin,
													  const float3					 rayDirection,
													  const float					 tMin,
													  const float					 tMax,
													  float3 *						 perRayData )
		{
			uint32_t p0, p1, p2;
			p0 = float_as_int( perRayData->x );
			p1 = float_as_int( perRayData->y );
			p2 = float_as_int( perRayData->z );

			optixTrace( th, // GAS
						rayOrigin,
						rayDirection,
						tMin,
						tMax,
						0.f, // ray time
						OptixVisibilityMask( 1 ),
						OPTIX_RAY_FLAG_NONE,
						0, // SBT offset
						0, // SBT stride
						0, // miss SBT index
						p0,
						p1,
						p2 );

			perRayData->x = int_as_float( p0 );
			perRayData->y = int_as_float( p1 );
			perRayData->z = int_as_float( p2 );
		}

		extern "C" __global__ void __raygen__()
		{
			const uint3 id	= optixGetLaunchIndex();
			const uint3 dim = optixGetLaunchDimensions();

			const Optix::RayGeneratorData * data
				= reinterpret_cast<Optix::RayGeneratorData *>( optixGetSbtDataPointer() );

			// camera data
			const float3 & origin = data->_camera._position;
			const float3 & front  = data->_camera._front;
			const float3 & du	  = data->_camera._du;
			const float3 & dv	  = data->_camera._dv;

			const float2 d = make_float2( float( id.x ) / float( dim.x ), float( id.y ) / float( dim.y ) ) * 2.f - 1.f;

			const float3 rayDir = normalize( du * d.x + dv * d.y + front );

			const uint32_t frameBufferId = id.x + id.y * dim.x;

			float3 normal;

			trace( params._traversable, // GAS
				   origin,
				   rayDir,
				   1e-3f, // tMin
				   1e16f, // tMax
				   &normal );

			params._frame._pixels[ frameBufferId ] = make_color( normal );
		}

		extern "C" __global__ void __intersection__sphere()
		{
			Optix::HitGroupData * data = reinterpret_cast<Optix::HitGroupData *>( optixGetSbtDataPointer() );

			// primitive data
			const int	   id	  = optixGetPrimitiveIndex();
			const float3 & center = data->_positions[ id ];
			const float	   radius = data->_radii[ id ];

			const float3 & o	 = optixGetObjectRayOrigin();
			const float3 & d	 = optixGetObjectRayDirection();
			const float3   oc	 = o - center;
			const float	   b	 = dot( oc, d );
			const float	   c	 = dot( oc, oc ) - radius * radius;
			const float	   delta = b * b - c;

			if ( delta > 0.f )
			{
				const float sqrtDelta = sqrtf( delta );

				float		t	 = ( -b - sqrtDelta ) ;
				const float tMin = optixGetRayTmin();
				const float tMax = optixGetRayTmax();
				if ( t <= tMax )
				{													// first intersection not too far
					if ( t < tMin ) { t = ( -b + sqrtDelta ); } // first intersection too near, check second one
					if ( t >= tMin && t <= tMax )					// t is within the interval
					{
						const float3 point	= o + d * t;
						const float3 normal = ( point - center ) / radius;

						unsigned int p0 = float_as_int( normal.x );
						unsigned int p1 = float_as_int( normal.y );
						unsigned int p2 = float_as_int( normal.z );

						optixReportIntersection( t, 0, p0, p1, p2 );
					}
				}
			}
		}
	} // namespace Renderer
} // namespace VTX
