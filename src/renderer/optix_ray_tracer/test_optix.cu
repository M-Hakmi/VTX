#include "hip/hip_runtime.h"

#include "cuda/random.hpp"
#include "optix_parameters.hpp"
#include <hip/device_functions.h>
#include <optix_device.h>

#define NB_PIXEL_SAMPLES 16

namespace VTX::Renderer::Optix
{
	using namespace CUDA;

	extern "C" __constant__ LaunchParameters params;

	static __forceinline__ __device__ void setPayload( const float3 & p )
	{
		optixSetPayload_0( float_as_int( p.x ) );
		optixSetPayload_1( float_as_int( p.y ) );
		optixSetPayload_2( float_as_int( p.z ) );
	}

	static __forceinline__ __device__ float3 getPayload()
	{
		return make_float3( int_as_float( optixGetPayload_0() ),
							int_as_float( optixGetPayload_1() ),
							int_as_float( optixGetPayload_2() ) );
	}

	__forceinline__ __device__ uchar4 make_color( const float3 & c )
	{
		return make_uchar4( static_cast<uint8_t>( clamp( c.x, 0.f, 1.f ) * 255.f ),
							static_cast<uint8_t>( clamp( c.y, 0.f, 1.f ) * 255.f ),
							static_cast<uint8_t>( clamp( c.z, 0.f, 1.f ) * 255.f ),
							255u );
	}

	extern "C" __global__ void __closesthit__sphere()
	{
		HitGroupData * data = reinterpret_cast<HitGroupData *>( optixGetSbtDataPointer() );

		const float3 normal = make_float3( int_as_float( optixGetAttribute_0() ),
										   int_as_float( optixGetAttribute_1() ),
										   int_as_float( optixGetAttribute_2() ) );
		// const int	   id		= optixGetPrimitiveIndex();
		const float3 & color	= params._colors[ 0 ]; // data->_spheres[ id ]._colorId ];
		const float3 & rayDir	= optixGetWorldRayDirection();
		const float	   radiance = fabsf( dot( rayDir, normal ) );

		setPayload( color * radiance );
	}

	extern "C" __global__ void __closesthit__cylinder()
	{
		HitGroupData * data = reinterpret_cast<HitGroupData *>( optixGetSbtDataPointer() );

		const float3 normal = make_float3( int_as_float( optixGetAttribute_0() ),
										   int_as_float( optixGetAttribute_1() ),
										   int_as_float( optixGetAttribute_2() ) );
		// const int	   id		= optixGetPrimitiveIndex();
		const float3 & color	= params._colors[ 0 ]; // data->_spheres[ id ]._colorId ];
		const float3 & rayDir	= optixGetWorldRayDirection();
		const float	   radiance = fabsf( dot( rayDir, normal ) );

		setPayload( color * radiance );
	}

	// extern "C" __global__ void __anyhit__() {}

	extern "C" __global__ void __miss__()
	{
		MissData * data	   = reinterpret_cast<MissData *>( optixGetSbtDataPointer() );
		float3	   payload = getPayload();
		setPayload( data->_colorBackground );
	}

	static __forceinline__ __device__ void trace( const OptixTraversableHandle & th,
												  const float3 &				 rayOrigin,
												  const float3 &				 rayDirection,
												  const float					 tMin,
												  const float					 tMax,
												  float3 *						 perRayData )
	{
		uint32_t p0, p1, p2;
		p0 = float_as_int( perRayData->x );
		p1 = float_as_int( perRayData->y );
		p2 = float_as_int( perRayData->z );

		optixTrace( th, // GAS
					rayOrigin,
					rayDirection,
					tMin,
					tMax,
					0.f, // ray time
					OptixVisibilityMask( 1 ),
					OPTIX_RAY_FLAG_NONE,
					0, // SBT offset
					0, // SBT stride
					0, // miss SBT index
					p0,
					p1,
					p2 );

		perRayData->x = int_as_float( p0 );
		perRayData->y = int_as_float( p1 );
		perRayData->z = int_as_float( p2 );
	}

	extern "C" __global__ void __raygen__()
	{
		const int idx  = optixGetLaunchIndex().x;
		const int idy  = optixGetLaunchIndex().y;
		const int dimx = optixGetLaunchDimensions().x;
		const int dimy = optixGetLaunchDimensions().y;

		const RayGeneratorData * data = reinterpret_cast<RayGeneratorData *>( optixGetSbtDataPointer() );

		// camera data
		const float3 & origin	 = data->_camera._position;
		const float3 & front	 = data->_camera._front;
		const float3 & du		 = data->_camera._du;
		const float3 & dv		 = data->_camera._dv;
		const int	   nbSamples = data->_nbSamples;

		uint32_t seed = tea<4>( idy * dimx + idx, 0 );

		float3 color;
		for ( int i = 0; i < nbSamples; ++i )
		{
			const float2 d
				= make_float2( ( idx + rnd( seed ) ) / float( dimx ), ( idy + rnd( seed ) ) / float( dimy ) ) * 2.f
				  - 1.f;
			float3		 c;
			const float3 rayDir = normalize( front + du * d.x + dv * d.y );

			trace( params._traversable, // GAS
				   origin,
				   rayDir,
				   1e-1f, // tMin
				   1e4f,  // tMax
				   &c );
			color.x += c.x;
			color.y += c.y;
			color.z += c.z;
		}

		const uint32_t frameBufferId		   = idx + idy * dimx;
		params._frame._pixels[ frameBufferId ] = make_color( color / nbSamples );
	}

	extern "C" __global__ void __intersection__sphere()
	{
		HitGroupData * data = reinterpret_cast<HitGroupData *>( optixGetSbtDataPointer() );

		// primitive data
		const int id = optixGetPrimitiveIndex();

		Intersection hit;
		if ( data->_spheres[ id ].intersect(
				 optixGetObjectRayOrigin(), optixGetObjectRayDirection(), optixGetRayTmin(), optixGetRayTmax(), hit ) )
		{
			unsigned int p0 = float_as_int( hit._normal.x );
			unsigned int p1 = float_as_int( hit._normal.y );
			unsigned int p2 = float_as_int( hit._normal.z );

			optixReportIntersection( hit._t, 0, p0, p1, p2, hit._colorId );
		}
	}

	extern "C" __global__ void __intersection__cylinder()
	{
		HitGroupData * data = reinterpret_cast<HitGroupData *>( optixGetSbtDataPointer() );

		// primitive data
		const int id = optixGetPrimitiveIndex();

		Intersection hit;
		if ( data->_cylinders[ id ].intersect(
				 optixGetObjectRayOrigin(), optixGetObjectRayDirection(), optixGetRayTmin(), optixGetRayTmax(), hit ) )
		{
			unsigned int p0 = float_as_int( hit._normal.x );
			unsigned int p1 = float_as_int( hit._normal.y );
			unsigned int p2 = float_as_int( hit._normal.z );

			optixReportIntersection( hit._t, 0, p0, p1, p2, hit._colorId );
		}
	}
} // namespace VTX::Renderer::Optix
